#include "hip/hip_runtime.h"
/*
 ============================================================================
 Author        : G. Barlas
 Version       : 1.0
 Last modified : December 2014
 License       : Released under the GNU GPL 3.0
 Description   :
 To build use  : make
 ============================================================================
 */
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <hip/hip_runtime.h>
#include <string.h>
#include "pgm.h"
#include "hip/hip_runtime.h"
#include ""
#include "hip/hip_runtime.h"

# define M_PI  3.14159265358979323846
const int degreeInc = 2;
const int degreeBins = 180 / degreeInc;
const int rBins = 100;
const float radInc = degreeInc * M_PI / 180;

//*****************************************************************

#define CUDA_CHECK_RETURN(value) {           \
    hipError_t _m_cudaStat = value;         \
    if (_m_cudaStat != hipSuccess) {        \
         fprintf(stderr, "Error %s at line %d in file %s\n",              \
                 hipGetErrorString(_m_cudaStat), __LINE__, __FILE__);    \
         exit(1);                                                         \
       } }


// The CPU function returns a pointer to the accummulator
void CPU_HoughTran(unsigned char* pic, int w, int h, int** acc)
{
    float rMax = sqrt(1.0 * w * w + 1.0 * h * h) / 2;  //(w^2 + h^2)/2, radio max equivalente a centro -> esquina
    *acc = new int[rBins * degreeBins];            //el acumulador, conteo depixeles encontrados, 90*180/degInc = 9000
    memset(*acc, 0, sizeof(int) * rBins * degreeBins); //init en ceros
    int xCent = w / 2;
    int yCent = h / 2;
    float rScale = 2 * rMax / rBins;

    for (int i = 0; i < w; i++) //por cada pixel
        for (int j = 0; j < h; j++) //...
        {
            int idx = j * w + i;
            if (pic[idx] > 0) //si pasa thresh, entonces lo marca
            {
                int xCoord = i - xCent;
                int yCoord = yCent - j;  // y-coord has to be reversed
                float theta = 0;         // actual angle
                for (int tIdx = 0; tIdx < degreeBins; tIdx++) //add 1 to all lines in that pixel
                {
                    float r = xCoord * cos(theta) + yCoord * sin(theta);
                    int rIdx = (r + rMax) / rScale;
                    (*acc)[rIdx * degreeBins + tIdx]++; //+1 para este radio r y este theta
                    theta += radInc;
                }
            }
        }
}

__constant__ float d_Cos[degreeBins];
__constant__ float d_Sin[degreeBins];

//*****************************************************************
//TODO Kernel memoria compartida
__global__ void GPU_HoughTranShared(unsigned char* pic, int w, int h, int* acc, float rMax, float rScale)
{
  int gloID = blockIdx.x * blockDim.x + threadIdx.x;
  if (gloID > w * h) return; // In case of extra threads in block
  
  int xCent = w / 2;
  int yCent = h / 2;

  int locID = threadIdx.x;

  int xCoord = gloID % w - xCent;
  int yCoord = yCent - gloID / w;

  // Use shared memory here for acc variable
  __shared__ int localAcc[degreeBins * rBins];
  // Initialize
  for (int i = locID; i < degreeBins * rBins; i += blockDim.x)
    localAcc[i] = 0;

  // warps sync
  __syncthreads ();

  if (pic[gloID] > 0)
  {
    for (int tIdx = 0; tIdx < degreeBins; tIdx++)
    {
      float r = xCoord * d_Cos[tIdx] + yCoord * d_Sin[tIdx];
      int rIdx = (r + rMax) / rScale;
      atomicAdd (localAcc + (rIdx * degreeBins + tIdx), 1);
    }
  }

  // warps sync again
  __syncthreads ();

  // Copy results from shared memory to global memory
  for (int i = locID ; i < degreeBins * rBins ; i += blockDim.x)
    atomicAdd (acc + i, localAcc[i]);

}


constexpr int top_size = 10;
void paintHough(unsigned char* pixels, int w, int h, int** results) 
{
    float rMax = sqrt(1.0 * w * w + 1.0 * h * h) / 2;
    int xCent = w / 2;
    int yCent = h / 2;
    float rScale = 2 * rMax / rBins;

    int top[top_size] = { -1 };

    for (int i = 0; i < w; i++) //por cada pixel
        for (int j = 0; j < h; j++) //...
        {
            int idx = j * w + i;
            if (pixels[idx] > 0) //si pasa thresh, entonces lo marca
            {
                int xCoord = i - xCent;
                int yCoord = yCent - j;  // y-coord has to be reversed
                float theta = 0;         // actual angle
                for (int tIdx = 0; tIdx < degreeBins; tIdx++) //add 1 to all lines in that pixel
                {
                    float r = xCoord * cos(theta) + yCoord * sin(theta);
                    int rIdx = (r + rMax) / rScale;
                    int current = (*results)[rIdx * degreeBins + tIdx]; 

                    //insertar en el top10
                     for (int topG = 0; topG < top_size; topG++)
                    {
                        if (top[topG] == current)
                            break;
                        if (top[topG] < current)
                        {
                            for (int rep = 9; rep > topG; rep--)
                                top[rep] = top[rep - 1];

                            top[topG] = current;
                            break;
                        }
                    }
                    theta += radInc;
                }
            }
        }

    for (int i = 0; i < w; i++) //por cada pixel
        for (int j = 0; j < h; j++) //...
        {
            int idx = j * w + i;

            int xCoord = i - xCent;
            int yCoord = yCent - j;  // y-coord has to be reversed
            float theta = 0;         // actual angle
            for (int tIdx = 0; tIdx < degreeBins; tIdx++) //add 1 to all lines in that pixel
            {
                float r = xCoord * cos(theta) + yCoord * sin(theta);
                int rIdx = (r + rMax) / rScale;
                int current = (*results)[rIdx * degreeBins + tIdx];
            
                //insertar en el top10
                for (int topG = 0; topG < top_size; topG++)
                    if (top[topG] == current)
                        pixels[idx] = pixels[idx] + 100 > 255 ? 255: pixels[idx] + 100;
                    
                theta += radInc;
            }
            
        }
}

//*****************************************************************
int main(int argc, char** argv)
{
    int i;

    PGMImage inImg(argv[1]);

    int* cpuht;
    int w = inImg.x_dim;
    int h = inImg.y_dim;

    hipEvent_t start, stop;
    float elapsedTime;

    int sharedMemorySize = sizeof(int) * rBins * degreeBins;

    // CPU calculation
    CPU_HoughTran(inImg.pixels, w, h, &cpuht);

    // pre-compute values to be stored
    float* pcCos = (float*)malloc(sizeof(float) * degreeBins);
    float* pcSin = (float*)malloc(sizeof(float) * degreeBins);
    float rad = 0;
    for (i = 0; i < degreeBins; i++)
    {
        pcCos[i] = cos(rad);
        pcSin[i] = sin(rad);
        rad += radInc;
    }

    float rMax = sqrt(1.0 * w * w + 1.0 * h * h) / 2;
    float rScale = 2 * rMax / rBins;

    // TODO eventualmente volver memoria global
    hipMemcpyToSymbol(HIP_SYMBOL(d_Cos), pcCos, sizeof(float) * degreeBins);
    hipMemcpyToSymbol(HIP_SYMBOL(d_Sin), pcSin, sizeof(float) * degreeBins);

    // setup and copy data from host to device
    unsigned char* d_in, * h_in;
    int* d_hough, * h_hough;

    h_in = inImg.pixels; // h_in contiene los pixeles de la imagen

    h_hough = (int*)malloc(degreeBins * rBins * sizeof(int));

    hipMalloc((void**)&d_in, sizeof(unsigned char) * w * h);
    hipMalloc((void**)&d_hough, sizeof(int) * degreeBins * rBins);
    hipMemcpy(d_in, h_in, sizeof(unsigned char) * w * h, hipMemcpyHostToDevice);
    hipMemset(d_hough, 0, sizeof(int) * degreeBins * rBins);

    // execution configuration uses a 1-D grid of 1-D blocks, each made of 256 threads
    //1 thread por pixel
    int blockNum = ceil(w * h / 256);

    CUDA_CHECK_RETURN(hipEventCreate(&start));
    CUDA_CHECK_RETURN(hipEventCreate(&stop));
    CUDA_CHECK_RETURN(hipEventRecord(start, 0));
    
    GPU_HoughTranShared<<<blockNum, 256>>>(d_in, w, h, d_hough, rMax, rScale);

    CUDA_CHECK_RETURN(hipEventRecord(stop, 0));
    CUDA_CHECK_RETURN(hipEventSynchronize(stop));
    CUDA_CHECK_RETURN(hipEventElapsedTime(&elapsedTime, start, stop));
    printf("Time: %.5f s\n", elapsedTime);

    // get results from device
    hipMemcpy(h_hough, d_hough, sizeof(int) * degreeBins * rBins, hipMemcpyDeviceToHost);

    // compare CPU and GPU results
    for (i = 0; i < degreeBins * rBins; i++)
    {
        if (cpuht[i] != h_hough[i])
            printf("Calculation mismatch at : %i %i %i\n", i, cpuht[i], h_hough[i]);
    }
    printf("Done!\n");

    paintHough(inImg.pixels, w, h, &h_hough);

    inImg.write("output.pgm");

    // TODO clean-up
    hipFree((void*)d_in);
    hipFree((void*)d_hough);
    hipFree((void*)d_Cos);
    hipFree((void*)d_Sin);

    free(pcCos);
    free(pcSin);
    free(h_hough);


    return 0;
}
